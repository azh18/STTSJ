#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "gpuKernel.h"

#include <stdio.h>

typedef struct GPUHausInfoTable {
	size_t *keywordNumP, *keywordNumQ; // #keywords in each point
	size_t taskNumP, taskNumQ; // #traj in each set
	size_t *pointNumP, *pointNumQ; // #points in each traj
};

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

size_t calculateDatasize_TrajSet(vector<STTraj> &trajSet) {
	size_t datasize = 0;
	for (vector<STTraj>::iterator it = trajSet.begin(); it != trajSet.end(); it++) {
		for (vector<STPoint>::iterator itp = it->points.begin(); itp != it->points.end(); itp++) {
			datasize += (2 * sizeof(double) + (itp->keywords.size()) * sizeof(int));
		}
	}
	return datasize;
}

// return the bytes copied, from pStart
// numKeywords: #keywords of each point
size_t copySTTrajToArray(STTraj &traj, char* pStart, size_t *numKeywords) {
	vector<STPoint>::iterator itp;
	size_t ptCnt = 0;
	char *s = pStart;
	for (itp = traj.points.begin(); itp != traj.points.end(); itp++) {
		numKeywords[ptCnt++] = itp->keywords.size();
		memcpy(pStart, &itp->lat, sizeof(double));
		pStart += sizeof(double);
		memcpy(pStart, &itp->lon, sizeof(double));
		pStart += sizeof(double);
		for (vector<int>::iterator itk = itp->keywords.begin(); itk != itp->keywords.end(); itk++) {
			memcpy(pStart, &(*itk), sizeof(int));
			pStart += sizeof(int);
		}
	}
	return (pStart - s);
}


int calculateDistanceGPU(vector<STTraj> trajSetP,
	vector<STTraj> trajSetQ,
	map<trajPair, double> &result) {
	char *dataSetP, *dataSetQ;
	size_t dataSizeP = 0, dataSizeQ = 0;
	// 所有点线性排列
	dataSizeP = calculateDatasize_TrajSet(trajSetP);
	dataSizeQ = calculateDatasize_TrajSet(trajSetQ);
	dataSetP = (char*)malloc(dataSizeP);
	dataSetQ = (char*)malloc(dataSizeQ);
	GPUHausInfoTable hausTaskInfo;
	

	vector<size_t> pointNumPCPU, pointNumQCPU,keywordNumPCPU, keywordNumQCPU;

	char *p = dataSetP, *q = dataSetQ;
	size_t copiedDataSize = 0;
	size_t keywordNum[1000];
	for (vector<STTraj>::iterator it = trajSetP.begin(); it != trajSetP.end(); it++) {
		copiedDataSize = copySTTrajToArray(*it, p, keywordNum);
		for (int i = 0; i < it->points.size(); i++) {
			keywordNumPCPU.push_back(keywordNum[i]);
		}
		pointNumPCPU.push_back(it->points.size());
		p = p + copiedDataSize;
	}
	for (vector<STTraj>::iterator it = trajSetQ.begin(); it != trajSetQ.end(); it++) {
		copiedDataSize = copySTTrajToArray(*it, q, keywordNum);
		for (int i = 0; i < it->points.size(); i++) {
			keywordNumQCPU.push_back(keywordNum[i]);
		}
		pointNumQCPU.push_back(it->points.size());
		q = q + copiedDataSize;
	}

	return 0;




}


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
 
/*
int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
*/

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
